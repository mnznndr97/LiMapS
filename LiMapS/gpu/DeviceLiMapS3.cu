#include "hip/hip_runtime.h"
﻿#include "DeviceLiMapSv3.cuh"

#include "cuda_shared.h"
#include <hip/hip_cooperative_groups.h>
#include <cuda/std/functional>
#include "cublas_shared.h"

#include "kernels.cuh"
#include "threshold_kernels.cuh"



static __device__ float* _solutionD;
static __device__ float* _signalD;
static __device__ float* _dictionaryD;
static __device__ float* _dictionaryInverseD;
static __device__ float* _alphaD;
static __device__ float* _alphaOldD;

static __device__ float* _beta;
static __device__ float* _intermD;

static __device__ float _signalSquareSum;
static __device__ float _alphaDiffSquareSum;

template<int unrollFactor>
__global__ void FillInterm(float* vector, size_t size) {
	int idx = blockIdx.x * (blockDim.x * unrollFactor) + threadIdx.x;

#pragma unroll
	for (size_t i = 0; i < unrollFactor; i++)
	{
		size_t vOffset = idx + i * blockDim.x;
		if (vOffset < size) vector[vOffset] = -_signalD[vOffset];
	}
}

template<int unrollFactor>
__global__ void FillAlpha(float* vector, size_t size) {
	int idx = blockIdx.x * (blockDim.x * unrollFactor) + threadIdx.x;

#pragma unroll
	for (size_t i = 0; i < unrollFactor; i++)
	{
		size_t vOffset = idx + i * blockDim.x;
		if (vOffset < size) vector[vOffset] = _beta[vOffset];
	}
}

template<int unrollFactor>
__global__ void ThresholdAlpha(float* vector, size_t size) {
	int idx = blockIdx.x * (blockDim.x * unrollFactor) + threadIdx.x;

#pragma unroll
	for (size_t i = 0; i < unrollFactor; i++)
	{
		size_t vOffset = idx + i * blockDim.x;
		if (vOffset < size) {
			if (fabs(_alphaD[vOffset]) < 1e-4f)
				_alphaD[vOffset] = 0.0f;
		}

	}
}

template<int unrollFactor>
__global__ void GetAlpha2(size_t dictionaryWords, size_t signalSize) {
	size_t idx = blockIdx.x * (blockDim.x * unrollFactor) + threadIdx.x;
	size_t idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idy >= dictionaryWords) return;

	float data = 0.0f;
#pragma unroll
	for (size_t i = 0; i < unrollFactor; i++)
	{
		size_t vOffset = idx + i * blockDim.x;
		float dicInverse = vOffset < signalSize ? _dictionaryInverseD[idy * signalSize + vOffset] : 0.0f;
		float signal = vOffset < signalSize ? _signalD[vOffset] : 0.0f;

		data += (dicInverse * signal);
	}

	KernelReduce<size_t>(data, signalSize, [](size_t index, float sum) {
		atomicAdd(&_alphaD[index], sum);
		atomicAdd(&_alphaOldD[index], sum);
		}, idy);
}

__global__ void CalculateBetaStep2(float lambda, size_t dictionaryWords, size_t signalSize) {
	cg::grid_group grid = cg::this_grid();

	unsigned long long index = grid.thread_rank();
	if (index >= dictionaryWords) {
		// Our thread is out of range
		return;
	}

	_beta[index] = GetBeta(lambda, _alphaD[index]);
}


template<int unrollFactor>
__global__ void CalculateIntermStep2(size_t dictionaryWords, size_t signalSize) {

	size_t idx = blockIdx.x * (blockDim.x * unrollFactor) + threadIdx.x;
	size_t idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idy >= signalSize) return;

	float data = 0.0f;
#pragma unroll
	for (size_t i = 0; i < unrollFactor; i++)
	{
		size_t vOffset = idx + i * blockDim.x;
		float dic = vOffset < dictionaryWords ? _dictionaryD[idy * dictionaryWords + vOffset] : 0.0f;
		float beta = vOffset < dictionaryWords ? _beta[vOffset] : 0.0f;

		data += (dic * beta);
		//data = fma(dic, beta, data);
	}

	KernelReduce<size_t>(data, dictionaryWords, [](size_t index, float sum) {
		atomicAdd(&_intermD[index], sum);
		}, idy);
}


template<int unrollFactor>
__global__ void CalculateNewAlphaStep2(size_t dictionaryWords, size_t signalSize) {
	size_t idx = blockIdx.x * (blockDim.x * unrollFactor) + threadIdx.x;
	size_t idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idy >= dictionaryWords) return;

	float data = 0.0f;
#pragma unroll
	for (size_t i = 0; i < unrollFactor; i++)
	{
		size_t vOffset = idx + i * blockDim.x;
		float dicInv = vOffset < signalSize ? _dictionaryInverseD[idy * signalSize + vOffset] : 0.0f;
		float interm = vOffset < signalSize ? _intermD[vOffset] : 0.0f;

		data += (dicInv * interm);
	}

	KernelReduce<size_t>(data, signalSize, [](size_t index, float sum) {
		atomicAdd(&_alphaD[index], -sum);
		}, idy);
}


__global__ void LiMapS2(size_t dictionaryWords, size_t signalSize) {
	// Handle to thread block group
	cg::grid_group grid = cg::this_grid();


	// 1) The first step of the LiMapS algorithm is to calculate the starting lamba coefficient. In order to do so, we need to calculate
	// the signal norm. So we enqueue on the default stream the SquareSum operation and then we wait for it.
	// The norm is foundamental for the next steps so there is nothing that we can do to avoid the sync time waste
	_signalSquareSum = 0.0f;
	dim3 blocks(256);
	SquareSumKrnlUnroll<8> << <GetGridSize(blocks, signalSize, 8), blocks, blocks.x / warpSize >> > (_signalD, signalSize, &_signalSquareSum);
	CUDA_CHECKD(hipDeviceSynchronize());

	assert(_signalSquareSum >= 0.0f);

	float t = sqrtf(_signalSquareSum);
	float lambda = 1.0f / t;

	_beta = new float[dictionaryWords];
	_intermD = new float[signalSize];

	blocks.x = 32;
	// 2) The second step of the algorithm is to prepare the starting alpha vector so also here we 
	// Launch the kernel calculation and we synchronize the device

	// Is it  necessary??
	//FillZero<1> << <gridSize, blocks >> > (_alphaD, dictionaryWords);
	//FillZero<1> << <gridSize, blocks >> > (_alphaOldD, dictionaryWords);

	dim3 gridSize = GetGridSize(blocks, signalSize, 8);
	gridSize.y = dictionaryWords;
	int sharedMemSize = blocks.x / warpSize;
	GetAlpha2<8> << <gridSize, blocks, sharedMemSize >> > (dictionaryWords, signalSize);
	CUDA_CHECKD(hipPeekAtLastError());
	CUDA_CHECKD(hipDeviceSynchronize());

	int i = 0;
	for (i = 0; i < 1000; i++)
	{
		// We set the alphaOld as the current alpha. We can do this by just swapping the pointer, avoiding 
		// useless data transfer
		cuda::std::swap(_alphaD, _alphaOldD);

		// From here, we split our computation next alpha computation in different step. This is necessary since some calculation
		// depend on data that should accessed after a global sync point (ex, after calculating the intermediate (dic * beta - sig) vector
		// Since global sync CANNOT be achieved (at least in old devices that not support grid_group::sync() method), we can do better:
		// we just queue our splitted work on the default stream, and then we just sync with the device at the end from this kenel.
		// In this way, the work is executed with all data dependencies respected

		// 3.1) We need to compute the beta vector for this iterarion
		blocks.x = 128;
		CalculateBetaStep2 << <GetGridSize(blocks, dictionaryWords), blocks >> > (lambda, dictionaryWords, signalSize);

		// 3.2) We need to compute the intermediate (dic * beta - sig) vector
		blocks.x = 64;
		FillInterm<8> << <GetGridSize(blocks, signalSize, 8), blocks >> > (_intermD, signalSize);

		blocks.x = 64;
		gridSize = GetGridSize(blocks, dictionaryWords, 8);
		gridSize.y = signalSize;
		int sharedMemSize = blocks.x / warpSize;
		CalculateIntermStep2<8> << <gridSize, blocks, sharedMemSize >> > (dictionaryWords, signalSize);
		CUDA_CHECKD(hipPeekAtLastError());

		blocks.x = 128;
		FillAlpha<8> << <GetGridSize(blocks, dictionaryWords, 8), blocks >> > (_alphaD, dictionaryWords);

		// 3.3) We compute the new alpha with the thresholding at the end
		blocks.x = 64;
		gridSize = GetGridSize(blocks, signalSize, 8);
		gridSize.y = dictionaryWords;
		sharedMemSize = blocks.x / warpSize;
		CalculateNewAlphaStep2<8> << <gridSize, blocks, sharedMemSize >> > (dictionaryWords, signalSize);

		blocks.x = 128;
		ThresholdAlpha<8> << <GetGridSize(blocks, dictionaryWords, 8), blocks >> > (_alphaD, dictionaryWords);

		lambda = 1.01f * lambda;

		// 3.4) We see how much alpha is changed
		_alphaDiffSquareSum = 0.0f;
		SquareDiffSumKrnlUnroll<8> << <GetGridSize(blocks, dictionaryWords, 8), blocks >> > (_alphaD, _alphaOldD, dictionaryWords, &_alphaDiffSquareSum);
		CUDA_CHECKD(hipDeviceSynchronize());

		float norm = sqrtf(_alphaDiffSquareSum);
		if (norm < 1e-5f) {
			break;
		}
	}

	printf("kernel iterations: %d\r\n", i);
	delete[] _beta;
	delete[] _intermD;
}

DeviceLiMapSv3::DeviceLiMapSv3(const float* solution, const float* signal, const float* D, const float* DINV, size_t dictionaryWords, size_t signalSize)
	: BaseLiMapS(solution, signal, D, DINV, dictionaryWords, signalSize)
{
	_alphaH.resize(_dictionaryWords);

	// We create the cuda pointers here and then we copy the pointers values to the device symbols. In this way
	// memory disposal should be automatically handled by the class
	_solutionPtr = make_cuda<float>(dictionaryWords);
	_signalPtr = make_cuda<float>(signalSize);
	_dictionaryPtr = make_cuda<float>(dictionaryWords * signalSize);
	_dictionaryInversePtr = make_cuda<float>(dictionaryWords * signalSize);
	_alphaPtr = make_cuda<float>(dictionaryWords);
	_alphaOldPtr = make_cuda<float>(dictionaryWords);

	float* dummyPtr = _solutionPtr.get();
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_solutionD), &dummyPtr, sizeof(void*)));

	dummyPtr = _signalPtr.get();
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_signalD), &dummyPtr, sizeof(void*)));

	dummyPtr = _dictionaryPtr.get();
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_dictionaryD), &dummyPtr, sizeof(void*)));

	dummyPtr = _dictionaryInversePtr.get();
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_dictionaryInverseD), &dummyPtr, sizeof(void*)));

	dummyPtr = _alphaPtr.get();
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_alphaD), &dummyPtr, sizeof(void*)));

	dummyPtr = _alphaOldPtr.get();
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_alphaOldD), &dummyPtr, sizeof(void*)));
}


void DeviceLiMapSv3::Execute(int iterations)
{
	CUDA_CHECK(hipMemcpyAsync(_signalPtr.get(), _signalHost, sizeof(float) * _signalSize, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpyAsync(_dictionaryInversePtr.get(), _dictionaryInverseHost, sizeof(float) * _dictionaryWords * _signalSize, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpyAsync(_dictionaryPtr.get(), _dictionaryHost, sizeof(float) * _dictionaryWords * _signalSize, hipMemcpyHostToDevice));

	LiMapS2 << < 1, 1 >> > (_dictionaryWords, _signalSize);

	CUDA_CHECK(hipMemcpyAsync(_alphaH.data(), _alphaPtr.get(), sizeof(float) * _dictionaryWords, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipDeviceSynchronize());
}
