#include "hip/hip_runtime.h"
﻿#include "kernels.cuh"

__global__ void SquareSumKrnl(const float* vec, size_t size, float* result) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// We calculate the squared value. We maintain the entire warp active but if we are out of bounds we
	// use zero as data value. In this way no sum error are introduced in the final sums
	float data = idx < size ? vec[idx] : 0.0f;
	data = data * data;

	KernelReduce<float*>(data, size, [](float* dest, float sum) { atomicAdd(dest, sum); }, result);
}

__global__ void Transpose(const float* __restrict__ source, float* destination, size_t width, size_t height) {
	// Handle to thread block group
	cg::thread_block cta = cg::this_thread_block();
	__shared__ float tile[TILE_DIM][TILE_DIM + 1];

	size_t size = width * height;

	int blockIdx_x, blockIdx_y;

	// do diagonal reordering
	if (width == height)
	{
		blockIdx_y = blockIdx.x;
		blockIdx_x = (blockIdx.x + blockIdx.y) % gridDim.x;
	}
	else
	{
		int bid = blockIdx.x + gridDim.x * blockIdx.y;
		blockIdx_y = bid % gridDim.y;
		blockIdx_x = ((bid / gridDim.y) + blockIdx_y) % gridDim.x;
	}

	// from here on the code is same as previous kernel except blockIdx_x replaces blockIdx.x
	// and similarly for y

	int xIndex = blockIdx_x * TILE_DIM + threadIdx.x;
	int yIndex = blockIdx_y * TILE_DIM + threadIdx.y;
	int index_in = xIndex + (yIndex)*width;

	// Let's read all the tile rows. Remember that each block reads 
	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
	{
		if (xIndex < width && yIndex < height)
			tile[threadIdx.y + i][threadIdx.x] = source[index_in + i * width];
	}

	cg::sync(cta);

	xIndex = blockIdx_y * TILE_DIM + threadIdx.x;
	yIndex = blockIdx_x * TILE_DIM + threadIdx.y;
	int index_out = xIndex + (yIndex)*height;

	for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
	{
		if (xIndex < height && yIndex < width)
			destination[index_out + i * height] = tile[threadIdx.x][threadIdx.y + i];
	}
}