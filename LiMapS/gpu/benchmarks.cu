#include "hip/hip_runtime.h"
﻿#include "benchmarks.cuh"

#include <iostream>

#include "cuda_shared.h"
#include "cublas_shared.h"
#include "kernels.cuh"
#include "threshold_kernels.cuh"


__global__ void Fill(float* data, size_t size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		data[index] = 1.0f;
	}
}

void RunCopyBenchmarks(size_t dataSize) {
	cuda_ptr<float> source = make_cuda<float>(dataSize);
	cuda_ptr<float> dest = make_cuda<float>(dataSize);

	hipblasHandle_t cublasHandle;
	/*CUBLAS_CHECK(hipblasCreate(&cublasHandle));
	hipblasScopy(cublasHandle, dataSize * sizeof(float), source.get(), 1, dest.get(), 1);
	CUBLAS_CHECK(hipblasDestroy(cublasHandle));*/

	dim3 blockSize(128);
	CopyTo<8> << <GetGridSize(blockSize, dataSize, 8), blockSize >> > (source.get(), dataSize, dest.get(), false);
	CUDA_CHECK(hipDeviceSynchronize());
}

void RunThresholdBenchmarks(size_t dataSize) {
	cuda_ptr<float> data = make_cuda<float>(dataSize);

	dim3 blockSize(256);
	dim3 gridSize((dataSize + blockSize.x - 1) / blockSize.x);
	hipMemset(data.get(), 0, dataSize * sizeof(float));

	std::cout << "Starting thresold kernel comparison benchmarks" << std::endl;

	blockSize.x = 32;
	gridSize.x = (dataSize + blockSize.x - 1) / blockSize.x;
	ThresholdVector<1> << <gridSize, blockSize.x >> > (data.get(), dataSize);
	CUDA_CHECK(hipDeviceSynchronize());

	ThresholdVectorAlwaysWrite<1> << <gridSize, blockSize.x >> > (data.get(), dataSize);
	CUDA_CHECK(hipDeviceSynchronize());

	ThresholdVector<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x >> > (data.get(), dataSize);
	CUDA_CHECK(hipDeviceSynchronize());

	ThresholdVectorAlwaysWrite<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x >> > (data.get(), dataSize);
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 128;
	ThresholdVector<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x >> > (data.get(), dataSize);
	CUDA_CHECK(hipDeviceSynchronize());

	ThresholdVectorAlwaysWrite<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x >> > (data.get(), dataSize);
	CUDA_CHECK(hipDeviceSynchronize());
}

void RunNormBenchmarks(size_t dataSize) {
	cuda_ptr<float> data = make_cuda<float>(dataSize);

	dim3 blockSize(256);
	dim3 gridSize((dataSize + blockSize.x - 1) / blockSize.x);
	Fill << <gridSize, blockSize.x >> > (data.get(), dataSize);

	float norm = 0.0f;
	std::cout << "Starting NORM kernel comparison benchmarks" << std::endl;
	hipblasHandle_t cublasHandle;
	CUBLAS_CHECK(hipblasCreate(&cublasHandle));
	hipblasSnrm2(cublasHandle, dataSize, data.get(), 1, &norm);
	CUBLAS_CHECK(hipblasDestroy(cublasHandle));

	std::cout << "Cublas norm: " << norm << std::endl;

	cuda_ptr<float> deviceNorm = make_cuda<float>(1);

	blockSize.x = 32;
	gridSize.x = (dataSize + blockSize.x - 1) / blockSize.x;
	SquareSumKrnl << <gridSize, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	hipMemcpy(&norm, deviceNorm.get(), sizeof(float), hipMemcpyDeviceToHost);
	norm = sqrt(norm);
	std::cout << "Norm from kernel: " << norm << std::endl;

	blockSize.x = 64;
	gridSize.x = (dataSize + blockSize.x - 1) / blockSize.x;
	SquareSumKrnl << <gridSize.x, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 128;
	SquareSumKrnl << <gridSize.x, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 256;
	SquareSumKrnl << <gridSize.x, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());


	// 2-reduction kernels
	hipMemset(deviceNorm.get(), CUBLAS_GEMM_ALGO0, sizeof(float));

	blockSize.x = 32;
	gridSize.x = (dataSize + blockSize.x - 1) / blockSize.x;
	SquareSumKrnlUnroll<2> << <(gridSize.x + 1) / 2, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	hipMemcpy(&norm, deviceNorm.get(), sizeof(float), hipMemcpyDeviceToHost);
	norm = sqrt(norm);
	std::cout << "Norm from kernel 2: " << norm << std::endl;

	blockSize.x = 64;
	gridSize.x = (dataSize + blockSize.x - 1) / blockSize.x;
	SquareSumKrnlUnroll<2> << <(gridSize.x + 1) / 2, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 128;
	SquareSumKrnlUnroll<2> << <(gridSize.x + 1) / 2, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 256;
	SquareSumKrnlUnroll<2> << <(gridSize.x + 1) / 2, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	// 8-reduction kernels
	hipMemset(deviceNorm.get(), CUBLAS_GEMM_ALGO0, sizeof(float));

	blockSize.x = 64;
	gridSize.x = (dataSize + blockSize.x - 1) / blockSize.x;
	gridSize.x = (gridSize.x + 7) / 8;
	SquareSumKrnlUnroll<8> << <gridSize.x, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	hipMemcpy(&norm, deviceNorm.get(), sizeof(float), hipMemcpyDeviceToHost);
	norm = sqrt(norm);
	std::cout << "Norm from kernel 8: " << norm << std::endl;

	blockSize.x = 128;
	gridSize.x = (dataSize + blockSize.x - 1) / blockSize.x;
	gridSize.x = (gridSize.x + 7) / 8;
	SquareSumKrnlUnroll<8> << <gridSize.x, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 256;
	gridSize.x = (dataSize + blockSize.x - 1) / blockSize.x;
	gridSize.x = (gridSize.x + 7) / 8;
	SquareSumKrnlUnroll<8> << <gridSize.x, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());



	// 16-reduction with cache kernels
	hipMemset(deviceNorm.get(), CUBLAS_GEMM_ALGO0, sizeof(float));

	blockSize.x = 128;
	SquareSumKrnlUnrollLdg<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	hipMemcpy(&norm, deviceNorm.get(), sizeof(float), hipMemcpyDeviceToHost);
	norm = sqrt(norm);
	std::cout << "Norm from kernel 8 - ldg: " << norm << std::endl;

	blockSize.x = 256;
	SquareSumKrnlUnrollLdg<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 512;
	SquareSumKrnlUnrollLdg<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());
}

void RunKernelsBenchmarks() {
	std::cout << "Starting benchmarks" << std::endl;

	size_t dataSize = 80000;
	cuda_ptr<float> zeroArray = make_cuda<float>(dataSize);
	cuda_ptr<float> destArray = make_cuda<float>(dataSize);

	dim3 blockSize(32);
	GetBetaKrnl << <(dataSize + blockSize.x - 1) / blockSize.x, blockSize.x >> > (1.0f, zeroArray.get(), destArray.get(), dataSize);
	hipDeviceSynchronize();

	blockSize.x = 64;
	GetBetaKrnl << <(dataSize + blockSize.x - 1) / blockSize.x, blockSize.x >> > (1.0f, zeroArray.get(), destArray.get(), dataSize);
	hipDeviceSynchronize();

	blockSize.x = 128;
	GetBetaKrnl << <(dataSize + blockSize.x - 1) / blockSize.x, blockSize.x >> > (1.0f, zeroArray.get(), destArray.get(), dataSize);
	hipDeviceSynchronize();

	blockSize.x = 256;
	GetBetaKrnl << <(dataSize + blockSize.x - 1) / blockSize.x, blockSize.x >> > (1.0f, zeroArray.get(), destArray.get(), dataSize);
	hipDeviceSynchronize();

	blockSize.x = 512;
	GetBetaKrnl << <(dataSize + blockSize.x - 1) / blockSize.x, blockSize.x >> > (1.0f, zeroArray.get(), destArray.get(), dataSize);
	hipDeviceSynchronize();
}