#include "hip/hip_runtime.h"
﻿#include "benchmarks.cuh"

#include <iostream>

#include "cuda_shared.h"
#include "cublas_shared.h"
#include "kernels.cuh"
#include "threshold_kernels.cuh"


__global__ void Fill(float* data, size_t size, float val = 1.0f) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		data[index] = val;
	}
}

void RunCopyBenchmarks(size_t dataSize) {
	cuda_ptr<float> source = make_cuda<float>(dataSize);
	cuda_ptr<float> dest = make_cuda<float>(dataSize);

	dim3 blockSize(128);
	CopyTo<8> << <GetGridSize(blockSize, dataSize, 8), blockSize >> > (source.get(), dataSize, dest.get(), false);
	CUDA_CHECK(hipDeviceSynchronize());
}

void RunThresholdBenchmarks(size_t dataSize) {
	cuda_ptr<float> data = make_cuda<float>(dataSize);

	dim3 blockSize(256);
	hipMemset(data.get(), 0, dataSize * sizeof(float));

	std::cout << "Starting thresold kernel comparison benchmarks" << std::endl;

	blockSize.x = 32;
	ThresholdVector<1> << <GetGridSize(blockSize, dataSize), blockSize.x >> > (data.get(), dataSize);
	CUDA_CHECK(hipDeviceSynchronize());

	ThresholdVectorAlwaysWrite<1> << <GetGridSize(blockSize, dataSize), blockSize.x >> > (data.get(), dataSize);
	CUDA_CHECK(hipDeviceSynchronize());

	ThresholdVector<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x >> > (data.get(), dataSize);
	CUDA_CHECK(hipDeviceSynchronize());

	ThresholdVectorAlwaysWrite<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x >> > (data.get(), dataSize);
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 128;
	ThresholdVector<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x >> > (data.get(), dataSize);
	CUDA_CHECK(hipDeviceSynchronize());

	ThresholdVectorAlwaysWrite<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x >> > (data.get(), dataSize);
	CUDA_CHECK(hipDeviceSynchronize());
}

void RunNormBenchmarks(size_t dataSize) {
	cuda_ptr<float> data = make_cuda<float>(dataSize);

	dim3 blockSize(256);
	Fill << <GetGridSize(blockSize, dataSize), blockSize.x >> > (data.get(), dataSize);

	float norm = 0.0f;
	std::cout << "Starting NORM kernel comparison benchmarks" << std::endl;
	hipblasHandle_t cublasHandle;
	CUBLAS_CHECK(hipblasCreate(&cublasHandle));
	hipblasSnrm2(cublasHandle, dataSize, data.get(), 1, &norm);
	CUBLAS_CHECK(hipblasDestroy(cublasHandle));

	std::cout << "Cublas norm: " << norm << std::endl;

	cuda_ptr<float> deviceNorm = make_cuda<float>(1);

	blockSize.x = 32;
	SquareSumKrnl << <GetGridSize(blockSize, dataSize), blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	hipMemcpy(&norm, deviceNorm.get(), sizeof(float), hipMemcpyDeviceToHost);
	norm = sqrt(norm);
	std::cout << "Norm from kernel: " << norm << std::endl;

	blockSize.x = 64;
	SquareSumKrnl << <GetGridSize(blockSize, dataSize), blockSize, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 128;
	SquareSumKrnl << <GetGridSize(blockSize, dataSize), blockSize, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 256;
	SquareSumKrnl << <GetGridSize(blockSize, dataSize), blockSize, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());


	// 2-reduction kernels
	hipMemset(deviceNorm.get(), 0, sizeof(float));

	blockSize.x = 32;
	SquareSumKrnlUnroll<2> << <GetGridSize(blockSize, dataSize, 2), blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	hipMemcpy(&norm, deviceNorm.get(), sizeof(float), hipMemcpyDeviceToHost);
	norm = sqrt(norm);
	std::cout << "Norm from kernel 2: " << norm << std::endl;

	blockSize.x = 64;
	SquareSumKrnlUnroll<2> << <GetGridSize(blockSize, dataSize, 2), blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 128;
	SquareSumKrnlUnroll<2> << <GetGridSize(blockSize, dataSize, 2), blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 256;
	SquareSumKrnlUnroll<2> << <GetGridSize(blockSize, dataSize, 2), blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	// 8-reduction kernels
	hipMemset(deviceNorm.get(), 0, sizeof(float));

	blockSize.x = 64;
	SquareSumKrnlUnroll<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	hipMemcpy(&norm, deviceNorm.get(), sizeof(float), hipMemcpyDeviceToHost);
	norm = sqrt(norm);
	std::cout << "Norm from kernel 8: " << norm << std::endl;

	blockSize.x = 128;
	SquareSumKrnlUnroll<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 256;
	SquareSumKrnlUnroll<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	// 16-reduction with cache kernels
	hipMemset(deviceNorm.get(), 0, sizeof(float));

	blockSize.x = 128;
	SquareSumGridUnroll<8> << <80, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	hipMemcpy(&norm, deviceNorm.get(), sizeof(float), hipMemcpyDeviceToHost);
	norm = sqrt(norm);
	std::cout << "Norm from kernel 8 - ldg: " << norm << std::endl;

	blockSize.x = 256;
	SquareSumGridUnroll<8> << <80, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 512;
	SquareSumGridUnroll<8> << <80, blockSize.x, blockSize.x / 32 >> > (data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());
}

void RunNormDiffBenchmarks(size_t dataSize) {
	cuda_ptr<float> data2 = make_cuda<float>(dataSize);
	cuda_ptr<float> data = make_cuda<float>(dataSize);

	dim3 blockSize(256);
	dim3 gridSize((dataSize + blockSize.x - 1) / blockSize.x);
	Fill << <gridSize, blockSize.x >> > (data.get(), dataSize);
	Fill << <gridSize, blockSize.x >> > (data2.get(), dataSize, 2.0f);

	cuda_ptr<float> deviceNorm = make_cuda<float>(1);
	hipMemset(deviceNorm.get(), 0, sizeof(float));

	// 8-reduction kernels
	blockSize.x = 64;
	SquareDiffSumKrnlUnroll<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x, blockSize.x / 32 >> > (data2.get(), data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	float norm = 0.0f;
	hipMemcpy(&norm, deviceNorm.get(), sizeof(float), hipMemcpyDeviceToHost);
	norm = sqrt(norm);
	std::cout << "NormDiff from kernel 8: " << norm << std::endl;

	blockSize.x = 128;
	SquareDiffSumKrnlUnroll<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x, blockSize.x / 32 >> > (data2.get(), data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 256;
	SquareDiffSumKrnlUnroll<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x, blockSize.x / 32 >> > (data2.get(), data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	// Cache streaming kernels
	hipMemset(deviceNorm.get(), 0, sizeof(float));

	blockSize.x = 128;
	SquareDiffSumKrnlUnrollLdg<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x, blockSize.x / 32 >> > (data2.get(), data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	hipMemcpy(&norm, deviceNorm.get(), sizeof(float), hipMemcpyDeviceToHost);
	norm = sqrt(norm);
	std::cout << "NormDiff from kernel 8 - ldg: " << norm << std::endl;

	blockSize.x = 256;
	SquareDiffSumKrnlUnrollLdg<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x, blockSize.x / 32 >> > (data2.get(), data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());

	blockSize.x = 512;
	SquareDiffSumKrnlUnrollLdg<8> << <GetGridSize(blockSize, dataSize, 8), blockSize.x, blockSize.x / 32 >> > (data2.get(), data.get(), dataSize, deviceNorm.get());
	CUDA_CHECK(hipDeviceSynchronize());
}


void RunKernelsBenchmarks() {
	std::cout << "Starting benchmarks" << std::endl;

	size_t dataSize = 80000;
	cuda_ptr<float> zeroArray = make_cuda<float>(dataSize);
	cuda_ptr<float> destArray = make_cuda<float>(dataSize);

	dim3 blockSize(32);
	GetBetaKrnl << <(dataSize + blockSize.x - 1) / blockSize.x, blockSize.x >> > (1.0f, zeroArray.get(), destArray.get(), dataSize);
	hipDeviceSynchronize();

	blockSize.x = 64;
	GetBetaKrnl << <(dataSize + blockSize.x - 1) / blockSize.x, blockSize.x >> > (1.0f, zeroArray.get(), destArray.get(), dataSize);
	hipDeviceSynchronize();

	blockSize.x = 128;
	GetBetaKrnl << <(dataSize + blockSize.x - 1) / blockSize.x, blockSize.x >> > (1.0f, zeroArray.get(), destArray.get(), dataSize);
	hipDeviceSynchronize();

	blockSize.x = 256;
	GetBetaKrnl << <(dataSize + blockSize.x - 1) / blockSize.x, blockSize.x >> > (1.0f, zeroArray.get(), destArray.get(), dataSize);
	hipDeviceSynchronize();

	blockSize.x = 512;
	GetBetaKrnl << <(dataSize + blockSize.x - 1) / blockSize.x, blockSize.x >> > (1.0f, zeroArray.get(), destArray.get(), dataSize);
	hipDeviceSynchronize();
}