#include "hip/hip_runtime.h"
﻿#include "DeviceLiMapSv2.cuh"

#include "cuda_shared.h"
#include <hip/hip_cooperative_groups.h>
#include "cublas_shared.h"

#include "kernels.cuh"
#include "beta_kernels.cuh"
#include "threshold_kernels.cuh"



__device__ float* _solutionD;
__device__ float* _signalD;
__device__ float* _dictionaryD;
__device__ float* _dictionaryInverseD;
__device__ float* _alphaD;
__device__ float* _alphaOldD;

__device__ float* _beta;
__device__ float* _intermD;

__device__ float _signalNorm;
__device__ float _alphaNorm;

__global__ void GetAlpha(size_t dictionaryWords, size_t signalSize) {
	cg::grid_group grid = cg::this_grid();
	if (grid.thread_rank() >= signalSize) {
		// Our thread is out of range
		return;
	}

	float sum = 0.0f;
	for (size_t i = 0; i < signalSize; i++)
	{
		sum = fmaf(_dictionaryInverseD[grid.thread_rank() * signalSize + i], _signalD[i], sum);
	}
	_alphaD[grid.thread_rank()] = sum;
}

__device__ void CalculateInterm(unsigned long long idx, size_t dictionaryWords, size_t signalSize) {
	if (idx >= signalSize) {
		// Our thread is out of range
		return;
	}

	float sum = 0.0f;
	for (size_t i = 0; i < dictionaryWords; i++)
	{
		sum += _dictionaryD[idx * dictionaryWords + i] * _beta[i];
		//sum = fmaf(_dictionaryD[idx * dictionaryWords + i], _beta[i], sum);
	}
	_intermD[idx] = sum - _signalD[idx];
}


__global__ void LiMapSImpl(float lambda, size_t dictionaryWords, size_t signalSize) {
	cg::grid_group grid = cg::this_grid();

	unsigned long long index = grid.thread_rank();
	if (index >= dictionaryWords) {
		// Our thread is out of range
		return;
	}

	float beta = GetBeta(lambda, _alphaD[index]);
	_beta[index] = beta;
	CUDA_CHECKD(hipDeviceSynchronize());

	CalculateInterm(index, dictionaryWords, signalSize);
	CUDA_CHECKD(hipDeviceSynchronize());

	float sum = 0.0f;
	for (size_t i = 0; i < signalSize; i++)
	{
		sum = fmaf(_dictionaryInverseD[index * signalSize + i], _intermD[i], sum);
	}
	float newAlpha = beta - sum;
	_alphaD[index] = newAlpha >= 1e-4f ? newAlpha : 0.0f;
}


__global__ void LiMapS(size_t dictionaryWords, size_t signalSize) {
	// Handle to thread block group
	cg::grid_group grid = cg::this_grid();

	_signalNorm = 0.0f;
	Norm(_signalD, signalSize, &_signalNorm);
	assert(_signalNorm >= 0.0f);

	float lambda = 1.0f / _signalNorm;

	_beta = new float[dictionaryWords];
	_intermD = new float[signalSize];

	dim3 blocks(256);
	dim3 gridSize((signalSize + blocks.x - 1) / blocks.x);
	GetAlpha << <gridSize, blocks >> > (dictionaryWords, signalSize);
	CUDA_CHECKD(hipDeviceSynchronize());
	memcpy(_alphaOldD, _alphaD, dictionaryWords * sizeof(float));

	int i = 0;
	for (i = 0; i < 1000; i++)
	{
		// We set the alphaOld as the current alpha
		float* temp;
		temp = _alphaD;
		_alphaD = _alphaOldD;
		_alphaOldD = temp;

		blocks.x = 32;
		gridSize.x = (dictionaryWords + blocks.x - 1) / blocks.x;
		LiMapSImpl << <gridSize, blocks >> > (lambda, dictionaryWords, signalSize);
		CUDA_CHECKD(hipDeviceSynchronize());

		lambda = 1.01f * lambda;
		NormDiff(_alphaD, _alphaOldD, dictionaryWords, &_alphaNorm);
		if (_alphaNorm < 1e-5f) {
			break;
		}
	}

	printf("kernel iterations: %d\r\n", i);
	delete[] _beta;
	delete[] _intermD;
}

DeviceLiMapSv2::DeviceLiMapSv2(std::vector<float>& solution, std::vector<float>& signal, std::vector<float>& D, std::vector<float>& DINV)
	:_signalSize(signal.size()), _dictionaryWords(solution.size()),
	// To avoid C++ vector copies, let's just store the vector references for our input data. This may be dangerous since the class MUST have the same (or shorted)
	// scope of our data, but for our purposes should be ok
	_hostSolution(solution), _hostSignal(signal), _hostDictionary(D), _hostDictionaryInverse(DINV)
{
	_solution = make_cuda<float>(solution.size());
	_signal = make_cuda<float>(signal.size());
	_dictionary = make_cuda<float>(D.size());
	_dictionaryInverse = make_cuda<float>(DINV.size());

	_alpha = make_cuda<float>(solution.size());
	_alphaOld = make_cuda<float>(solution.size());

	// We copy the 
	float* dummyPtr = _solution.get();
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_solutionD), &dummyPtr, sizeof(void*)));

	dummyPtr = _signal.get();
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_signalD), &dummyPtr, sizeof(void*)));

	dummyPtr = _dictionary.get();
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_dictionaryD), &dummyPtr, sizeof(void*)));

	dummyPtr = _dictionaryInverse.get();
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_dictionaryInverseD), &dummyPtr, sizeof(void*)));

	dummyPtr = _alpha.get();
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_alphaD), &dummyPtr, sizeof(void*)));

	dummyPtr = _alphaOld.get();
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(_alphaOldD), &dummyPtr, sizeof(void*)));
}

DeviceLiMapSv2::~DeviceLiMapSv2()
{

}

void DeviceLiMapSv2::Execute(int iterations)
{
	CUDA_CHECK(hipMemcpy(_signal.get(), _hostSignal.data(), sizeof(float) * _signalSize, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(_dictionaryInverse.get(), _hostDictionaryInverse.data(), sizeof(float) * _dictionaryWords * _signalSize, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(_dictionary.get(), _hostDictionary.data(), sizeof(float) * _dictionaryWords * _signalSize, hipMemcpyHostToDevice));

	dim3 blocks(32);
	dim3 gridSize((_dictionaryWords + blocks.x - 1) / blocks.x);

	//LiMapS << < gridSize, blocks >> > (_dictionaryWords, _signalSize);
	LiMapS << < 1, 1 >> > (_dictionaryWords, _signalSize);
	CUDA_CHECK(hipDeviceSynchronize());
	/*

	CUBLAS_CHECK(hipblasSetVector(_signalSize, sizeof(float), _hostSignal.data(), 1, _signal.get(), 1));
	CUDA_CHECK(hipMemcpy(_dictionaryInverse.get(), _hostDictionaryInverse.data(), sizeof(float) * _dictionaryWords * _signalSize, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(_dictionary.get(), _hostDictionary.data(), sizeof(float) * _dictionaryWords * _signalSize, hipMemcpyHostToDevice));

	const float alphaScalar = 1.0f;
	const float negAlphaScalar = -1.0f;
	const float betaScalar = 0.0f;
	CUBLAS_CHECK(hipblasSgemv(_cublasHandle, HIPBLAS_OP_T, _signalSize, _dictionaryWords, &alphaScalar, _dictionaryInverse.get(), _signalSize, _signal.get(), 1, &betaScalar, _alpha.get(), 1));

	float signalNorm = 0.0f;
	CUBLAS_CHECK(hipblasSnrm2(_cublasHandle, _signalSize, _signal.get(), 1, &signalNorm));
	float lambda = 1.0f / signalNorm;

	cuda_ptr<float> beta = make_cuda<float>(_dictionaryWords);
	cuda_ptr<float> interm = make_cuda<float>(_signalSize);

	dim3 blockSize(128);
	dim3 gridSize(_dictionaryWords + blockSize.x - 1 / blockSize.x);

	int iteration = 0;
	for (; iteration < iterations; iteration++)
	{
		// First we save the current alpha as the old one, in order to use it later
		CUBLAS_CHECK(hipblasScopy(_cublasHandle, _dictionaryWords, _alpha.get(), 1, _alphaOld.get(), 1));

		GetBetaKrnl << <gridSize, blockSize >> > (lambda, _alpha.get(), beta.get(), _dictionaryWords);

		CUBLAS_CHECK(hipblasSgemv(_cublasHandle, HIPBLAS_OP_T, _dictionaryWords, _signalSize, &alphaScalar, _dictionary.get(), _dictionaryWords, beta.get(), 1, &betaScalar, interm.get(), 1));
		CUBLAS_CHECK(hipblasSaxpy(_cublasHandle, _signalSize, &negAlphaScalar, _signal.get(), 1, interm.get(), 1));

		CUBLAS_CHECK(hipblasSgemv(_cublasHandle, HIPBLAS_OP_T, _signalSize, _dictionaryWords, &alphaScalar, _dictionaryInverse.get(), _signalSize, interm.get(), 1, &betaScalar, _alpha.get(), 1));
		// axpy takes a single input/out parameter so we have to negate our subtraction and then multiply the result by -1.0 later
		CUBLAS_CHECK(hipblasSaxpy(_cublasHandle, _dictionaryWords, &negAlphaScalar, beta.get(), 1, _alpha.get(), 1));
		CUBLAS_CHECK(hipblasSscal(_cublasHandle, _dictionaryWords, &negAlphaScalar, _alpha.get(), 1));

		ThresholdKrnl << <gridSize, blockSize >> > (_alpha.get(), _dictionaryWords, _alphaElementTh);
		CUBLAS_CHECK(hipblasSaxpy(_cublasHandle, _dictionaryWords, &negAlphaScalar, _alpha.get(), 1, _alphaOld.get(), 1));

		lambda = lambda * gamma;

		float diffNorm = 0.0f;
		CUBLAS_CHECK(hipblasSnrm2(_cublasHandle, _dictionaryWords, _alphaOld.get(), 1, &diffNorm));
		if (diffNorm < _epsilon) {
			// We are done with the iterations. Norm is very small
			break;
		}
	}

	*/
}
