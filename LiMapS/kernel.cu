#include "hip/hip_runtime.h"
﻿#include<iostream>
#include <vector>
#include <random>
#include <assert.h>
#include <fstream>
#include <string>
#include <sstream>

#include "cpu/vectors.hpp"
#include "cpu/matrices.hpp"

template <class T>
void ReadColumnVector(const std::string& file, std::vector<T>& dest) {
	std::ifstream stream(file);
	assert(stream.is_open());

	// Actual solution is a column vector, in the matrix file each element is on a single line
	T data;
	while (stream >> data) {
		dest.push_back(data);
	}
}

void ReadMatrix(const std::string& file, std::unique_ptr<float[]>& dest, size_t rows, size_t cols) {
	std::ifstream stream(file);
	assert(stream.is_open());

	std::string line;
	size_t rowIndex = 0;

	while (std::getline(stream, line))
	{
		std::istringstream lineStream(line);
		std::string floatData;
		size_t colIndex = 0;
		while (std::getline(lineStream, floatData, ','))
		{
			dest[rowIndex * cols + colIndex] = std::stof(floatData);
			++colIndex;
		}
		assert(colIndex == cols);

		++rowIndex;
	}
	assert(rowIndex == rows);

}

int main()
{
	std::cout << " *** LiMapS Implementation ***" << std::endl;

	const int signalSize = 200;
	const int dictionaryWords = 800;

	std::vector<float> actualSolution;
	std::vector<float> signal;
	std::unique_ptr<float[]> dictionary = std::make_unique<float[]>(signalSize * dictionaryWords);
	std::unique_ptr<float[]> dictionaryInverse = std::make_unique<float[]>(signalSize * dictionaryWords);

	// Let' s read our data from a file for the moment and assert that evertything has the right dimension
	ReadColumnVector("data\\1\\in_true_alpha.txt", actualSolution);
	ReadColumnVector("data\\1\\in_signal.txt", signal);
	ReadMatrix("data\\1\\in_D.txt", dictionary, signalSize, dictionaryWords);
	ReadMatrix("data\\1\\in_D_inverse.txt", dictionaryInverse, dictionaryWords, signalSize);

	assert(actualSolution.size() == dictionaryWords);
	assert(signal.size() == signalSize);

	std::cout << "# Dictionary atoms: " << dictionaryWords << std::endl;
	std::cout << "Signal size: " << signalSize << std::endl;

	// Stopping criteria declaration
	const float epsilon = 1e-5;
	const float alpha_min = 1e-4;
	const int maxIterations = 1000;

	std::vector<float> alpha(dictionaryWords, 0.0f);
	std::vector<float> alpha_old(dictionaryWords, 0.0f);

	Mat2VecProduct(dictionaryInverse.get(), dictionaryWords, signalSize, signal.data(), signalSize, alpha.data());

	float signalNorm = GetEuclideanNorm(signal);
	// Just temporary assert to be sure we are doing ok 
	assert(abs(signalNorm - 129.0749) < 0.0001f);

	float lambda = 1.0f / signalNorm;
	float gamma = 1.01f;

	for (int i = 0; i < maxIterations; i++)
	{
		
	}

	return 0;
}

